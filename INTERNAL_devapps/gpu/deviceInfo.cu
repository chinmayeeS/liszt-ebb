#include "stdio.h"

#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


int main (int argc, char **argv) {
	int devID;
	hipDeviceProp_t devProp;

	devID = findCudaDevice(argc, (const char**) argv);
	checkCudaErrors(hipGetDeviceProperties(&devProp, devID));

	printf("warpSize : %d\n",                    devProp.warpSize);
	printf("maxThreadsPerBlock : %d\n",          devProp.maxThreadsPerBlock);
	printf("maxThreadsPerMultiprocessor : %d\n", devProp.maxThreadsPerMultiProcessor);
	printf("sharedMemPerBlock : %lu\n",          devProp.sharedMemPerBlock);
	printf("regsPerBlock : %d\n",                devProp.regsPerBlock);
	printf("multiProcessorCount : %d\n",         devProp.multiProcessorCount);
}